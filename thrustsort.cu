#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>

  //compile with:
  // nvcc -m64 -arch=sm_35 thrustsort.cu -lcudart -O3 -o thrustsort
 // nvcc  thrustsort.cu -o thrustsort

int main(void)
{
    hipSetDevice (0);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // generate 32M random numbers serially
    thrust::host_vector<int> h_vec(16 << 20);
    std::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    float elapsedTime;
    hipDeviceSynchronize();
    hipEventRecord(start, 0);

    // sort data on the device (846M keys per second on GeForce GTX 480)
    thrust::sort(d_vec.begin(), d_vec.end());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("Processing time: %f (ms)\n", elapsedTime);

    return 0;
}
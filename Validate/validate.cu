
// nvcc -m64 -arch=sm_35 validate.cu -lcudart -O3 -o validate
// nvcc validate.cu -o validate ; ./validate 20 1 0


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "helper.h"
// #include "sort.cu"


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------
#define SIZE 8
bool    g_verbose = false;  // Whether to display input/output to console
int     num_items = SIZE;
int     deviceid = 0;

// MAIN
int main (int argc, char** argv){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    argsHandler (argc, argv, &num_items, &g_verbose, &deviceid);

    hipSetDevice (deviceid);

    // Discription
    printf("Sorting %d items (%d-byte keys %d-byte values)\n",
        num_items, int(sizeof(float)), int(sizeof(int)));
    fflush(stdout);

    // Allocate host arrays
    float*      h_keys             = new float[num_items];
    float*      h_reference_keys   = new float[num_items];
    int*        h_values           = new int[num_items];
    int*        h_reference_values = new int[num_items];

    // Allocate host arrays
    float*      d_keys;
    int*        d_values;
    CUDA_SAFE_CALL(hipMallocManaged(&d_keys, sizeof(float)));
    CUDA_SAFE_CALL(hipMallocManaged(&d_values, sizeof(int)));

    // Initialize problem and solution on host
    Initialize(h_keys, h_values, h_reference_keys, h_reference_values, num_items, g_verbose);

    // Copy the data to the device
    hipMemcpy(d_keys, h_keys,  sizeof(float) * num_items, hipMemcpyHostToDevice);
    hipMemcpy(d_values, h_values, sizeof(int) * num_items, hipMemcpyHostToDevice);

    // Start timer
    float elapsedTime;
    hipDeviceSynchronize();
    hipEventRecord(start, 0);

    // Run the program or Kernel
    // sort(d_keys, d_values, num_items);
    // sortkernel <<blocks,threads>>(d_keys, d_values, num_items); 

    // Stop timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Processing time: %f (ms)\n", elapsedTime);

    // Copy the data back to host
    hipMemcpy(h_keys, d_keys,  sizeof(float) * num_items, hipMemcpyDeviceToHost);
    hipMemcpy(h_values, d_values, sizeof(int) * num_items, hipMemcpyDeviceToHost);

    // just for test remove these for actual run (cheating)
    //*************************
    memcpy(h_keys, h_reference_keys, sizeof(float) * num_items);
    memcpy(h_values, h_reference_values, sizeof(int) * num_items);
    //**************************

     if (g_verbose){
        printf("Computed keys: \n");
        DisplayResults(h_keys, num_items);
        printf("\n\n");
        printf("Computed values: \n");
        DisplayResults(h_values, num_items);
        printf("\n\n");
    }

    // Check for correctness (and display results, if specified)
    int compare;
    compare = CompareResults(h_keys, h_reference_keys, num_items, g_verbose);
    printf("\t Compare keys: %s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    compare = CompareResults(h_values, h_reference_values , num_items, g_verbose);
    printf("\t Compare values: %s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

   

    double dTimeSecs = 1.0e-3 * elapsedTime ;
    printf("Sorting Network, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u elements, NumDevsUsed = %u\n",
    (1.0e-6 * (double)num_items/dTimeSecs), dTimeSecs , num_items, 1);

    // Cleanup
    if (h_keys) delete[] h_keys;
    if (h_reference_keys) delete[] h_reference_keys;
    if (h_values) delete[] h_values;
    if (h_reference_values) delete[] h_reference_values;
    if (d_keys) CUDA_SAFE_CALL(hipFree(d_keys));
    if (d_values) CUDA_SAFE_CALL(hipFree(d_values));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
}
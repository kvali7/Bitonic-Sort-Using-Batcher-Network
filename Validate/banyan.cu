#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdio.h>
#include <thrust/swap.h>
using namespace std;

// HOST helper function: get N given size of list
int getN(int size) {
  int N = 2; 
  while (N < size) {
    N = N*2;
  }
  return N;
}

// SHUFFLE kernel for N=pow(2,n) elements
__global__
void shuffleN(float *in, float *out, int size, ulong N) {
  // TO-DO: pull in shared memory
  // unsigned long ind_in  = threadIdx.x; // thread within block
  // unsigned long ind_in_base = blockIdx.x*blockDim.x; // block within col
  unsigned long ind_in_base = blockIdx.x*size; // block within col; accommodate fixed max threadNum=1024
  unsigned long ind_out; 
  unsigned long stride = blockDim.x; 
  for (unsigned long ind_in = threadIdx.x; ind_in < (unsigned long) size; ind_in += stride) {
    // ind_out = 2*(ind_in-ind_in % size/2);
    if (ind_in % 2 == 0) { // even 
      ind_out = ind_in/2;
    }  
    else {  // odd 
      ind_out = (size/2)+(ind_in/2);
    }
    out[ind_in_base+ind_in]=in[ind_in_base+ind_out];
    // using "out" as ping-pong; write back to "in"
    // __syncthreads();
    // in[ind_in_base+ind_in]=out[ind_in_base+ind_in];
  }
  __syncthreads();
  for (unsigned long ind_in = threadIdx.x; ind_in < (unsigned long) size; ind_in += stride) {
    in[ind_in_base+ind_in]=out[ind_in_base+ind_in];
  }
}


// BUTTERFLY kernel for N=pow(2,n) elements
__global__
void butterflyN(float *in, float *out, int size, ulong N) {
  // TO-DO: pull in shared memory
  unsigned long ind_in_base = blockIdx.x*size; // block within col; accommodate fixed max threadNum=1024
  unsigned long ind_out; 
  unsigned long stride = blockDim.x; 
  for (unsigned long ind_in = threadIdx.x; ind_in < (unsigned long) size; ind_in += stride) {
    if (ind_in < size/2) { // first half of list
      if (ind_in%2==0) // even
        ind_out = ind_in;
      else // odd
        ind_out = ind_in + (size/2 - 1);
    }  
    else {  // second half of list
      if (ind_in%2==0) // even
        ind_out = ind_in - (size/2 - 1);
      else //odd
        ind_out = ind_in;
    }
    out[ind_in_base+ind_in]=in[ind_in_base+ind_out];
  }
  // using "out" as ping-pong; write back to "in"
  __syncthreads();
  for (unsigned long ind_in = threadIdx.x; ind_in < (unsigned long) size; ind_in += stride) {
    in[ind_in_base+ind_in]=out[ind_in_base+ind_in];
  }
}

// COMPARE AND SWAP 
__global__
void compareAndSwap(float *in, int level, ulong N) {
  // TO-DO: pull in shared memory
  unsigned long stride = blockDim.x*gridDim.x; 
  // if (comp_ind < N) {
  for (unsigned long comp_ind = (unsigned long) blockDim.x*blockIdx.x+threadIdx.x;
       comp_ind < N/2;
       comp_ind += stride) {
    // STEP 0: Get direction of comparison
    bool comp_bool = (bool)(comp_ind & (0x1 << level)); 
    // STEP 1: Write out results of comparisons
    unsigned long data0 = 2*comp_ind;
    unsigned long data1 = 2*comp_ind+1;
    // compare and swap based on comp_bool
    if ( (comp_bool && in[data0] < in[data1]) || (!comp_bool && in[data0] >= in[data1]) ) { 
        thrust::swap(*(in+data0), *(in+data1)); 
    }
  }
}

void banyan(float *x, ulong N, uint n) {
  // bitonic mergesort on batcher-banyan network
  float*       y;
  CUDA_SAFE_CALL(hipMallocManaged(&y, N * sizeof(float)));
  int level = 0;
  int stage = 0;
  int substage = 0;
  int div; // blockNum for current routing kernel
  int threadNum; // threadNum for current routing kernel
  int compThreadNum = 512; // threadNum for compareAndSwap kernel
  int compBlockNum = min((long)65535,(N+compThreadNum-1)/compThreadNum); // max(blockNum)=65535

  while (stage < n) {
    while (substage <= stage) { 
      div = N/(pow(2,2+stage-substage));
      // printf("stage=%d - substage=%d - div=%d - level=%d\n", stage, substage, div, level);
      if (stage < n-1) {
        threadNum = min((long)1024, N/div); 
        if (substage == 0) {
          compareAndSwap<<<compBlockNum,compThreadNum>>>(x, level, N);
          hipDeviceSynchronize();
          // printf("-> compare for stage=%d at level=%d\n", stage, level);
          shuffleN<<<div,threadNum>>>(x, y, N/div, N);
          hipDeviceSynchronize();
          // printf("-> shuffle for stage=%d\n", stage);
          level++;
        }
        compareAndSwap<<<compBlockNum,compThreadNum>>>(x, level, N);
        hipDeviceSynchronize();
        // printf("-> compare for stage=%d at level=%d\n", stage, level);
        butterflyN<<<div,threadNum>>>(x, y, N/div, N);
        hipDeviceSynchronize();
        // printf("-> butterfly for stage=%d, substage=%d\n", stage, substage);
        substage++;
      }
      else {
        compareAndSwap<<<compBlockNum,compThreadNum>>>(x, level, N);
        hipDeviceSynchronize();
        // printf("-> final compare for stage=%d at level=%d\n", stage, level);
        break;
      }
    }
    stage++;
    substage = 0;
  }


  if (y) CUDA_SAFE_CALL(hipFree(y));

  return;
}

#include "hip/hip_runtime.h"

// nvcc -m64 -arch=sm_35 validate_banyan.cu -lcudart -O3 -o validate_banyan
// nvcc validate_banyan.cu -o validate_banyan ; ./validate_banyan 16 1 0

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "helper_nov.h"
#include "banyan.cu"
using namespace std;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------
#define SIZE 8
bool    g_verbose = false;  // Whether to display input/output to console
ulong     num_items = SIZE;
int     deviceid = 0;

// MAIN
int main (int argc, char** argv){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    argsHandler (argc, argv, &num_items, &g_verbose, &deviceid);

    ulong N = num_items;  
    if (!IsPowerOfTwo(N)){
        fprintf(stderr, "Numberof items is not a power of two"
        "\n");
        exit(1);  
    }
    uint n = log2((double)N); // n is log2 of N
    hipSetDevice (deviceid);

    // Discription
    printf("Sorting %d items (%d-byte keys) using Banyan Network, %d total stages\n",
        N, int(sizeof(float)), n);
    fflush(stdout);

    // Allocate host arrays
    float*      h_keys             = new float[N];
    float*      h_reference_keys   = new float[N];

    // Allocate device arrays
    // copied from benyan.cu
    float*       x;
    float*       y;
    bool*        comparators; 
    CUDA_SAFE_CALL(hipMallocManaged(&x, N * sizeof(float)));
    CUDA_SAFE_CALL(hipMallocManaged(&y, N * sizeof(float)));
    CUDA_SAFE_CALL(hipMallocManaged(&comparators, N/2 * sizeof(bool)));

    // Initialize problem and solution on host
    Initialize(h_keys, h_reference_keys, N, g_verbose);

    // Copy the data to the device
    hipMemcpy(x, h_keys,  sizeof(float) * N, hipMemcpyHostToDevice);

    // Start timer
    float elapsedTime;
    hipDeviceSynchronize();
    hipEventRecord(start, 0);

    // Run the program or Kernel
    benyan(x, y, comparators , N, n);

    // Stop timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Processing time: %f (ms)\n", elapsedTime);

    // Copy the data back to host
    hipMemcpy(h_keys, x,  sizeof(float) * N, hipMemcpyDeviceToHost);

    // just for test remove these for actual run (cheating)
    //*************************
    // memcpy(h_keys, h_reference_keys, sizeof(float) * N);
    //**************************


     if (g_verbose){
        printf("Computed keys: \n");
        DisplayResults(h_keys, N);
        printf("\n\n");
    }

    // Check for correctness (and display results, if specified)
    int compare;
    compare = CompareResults(h_keys, h_reference_keys, N, g_verbose);
    printf("\t Compare keys: %s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

   

    double dTimeSecs = 1.0e-3 * elapsedTime ;
    printf("Sorting Network, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u elements, NumDevsUsed = %u\n",
    (1.0e-6 * (double)N/dTimeSecs), dTimeSecs , N, 1);

    // Cleanup
    if (h_keys) delete[] h_keys;
    if (h_reference_keys) delete[] h_reference_keys;
    if (x) CUDA_SAFE_CALL(hipFree(x));
    if (y) CUDA_SAFE_CALL(hipFree(y));
    if (comparators) CUDA_SAFE_CALL(hipFree(comparators));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
}
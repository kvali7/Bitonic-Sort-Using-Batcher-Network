#include "hip/hip_runtime.h"
// nvcc -m64 -arch=sm_35 validate_banyan.cu -lcudart -O3 -o validate_banyan
// nvcc validate_banyan.cu -o validate_banyan ; ./validate_banyan 16 1 0

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "helper_nov.h"
#include "banyan.cu"
using namespace std;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------
#define SIZE 16
bool    g_verbose = false;  // Whether to display input/output to console
ulong     num_items = SIZE;
int     deviceid = 0;
ulong N;

// MAIN
int main (int argc, char** argv){

    hipSetDevice (deviceid);

    int minn = 4;
    int maxn = 26;
    int maxiter = 8;
    double time[30] = {0};
    double speed[30] = {0};
    double timetemp[10] = {0};
    double speedtemp[10] = {0};
    double timesum = 0;
    double speedsum = 0;

    int n = 24;
    for ( n = minn; n <= maxn; n++){
      N = pow(2,n);
      memset (timetemp, 0, 10* sizeof(double));
      memset (speedtemp, 0 , 10* sizeof(double));
      for (int iter = 0; iter < maxiter; iter++){

            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            // argsHandler (argc, argv, &num_items, &g_verbose, &deviceid);

            // ulong N = num_items;  
            if (!IsPowerOfTwo(N)){
                fprintf(stderr, "Numberof items is not a power of two"
                "\n");
                exit(1);  
            }
            // uint n = log2((double)N); // n is log2 of N

            // Discription
            printf("Sorting %d items (%d-byte keys) using Banyan Network, %d total stages\n",
                N, int(sizeof(float)), n);
            printf("banyan_batcher in function call: N=%d - n=%d (sorting %d-byte keys) \n",(int)N,(int)n, int(sizeof(float)));

            fflush(stdout);

            // Allocate host arrays
            float*      h_data             = new float[N];
            float*      h_reference_data   = new float[N];

            // Allocate device arrays
            // copied from banyan.cu
            float*       d_data;
            CUDA_SAFE_CALL(hipMallocManaged(&d_data, N * sizeof(float)));

            // Initialize problem and solution on host
            Initialize(h_data, h_reference_data, N, g_verbose);

            // Copy the data to the device
            hipMemcpy(d_data, h_data,  sizeof(float) * N, hipMemcpyHostToDevice);

            // Start timer
            float elapsedTime;
            hipDeviceSynchronize();
            hipEventRecord(start, 0);

            // Run the program or Kernel
            banyan(d_data , N, n);

            // Stop timer
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("Processing time: %f (ms)\n", elapsedTime);

            // Copy the data back to host
            hipMemcpy(h_data, d_data,  sizeof(float) * N, hipMemcpyDeviceToHost);

            // just for test remove these for actual run (cheating)
            //*************************
            // memcpy(h_data, h_reference_data, sizeof(float) * N);
            //**************************


            if (g_verbose){
                printf("Computed keys: \n");
                DisplayResults(h_data, N);
                printf("\n\n");
            }

            // Check for correctness (and display results, if specified)
            int compare;
            compare = CompareResults(h_data, h_reference_data, N, g_verbose);
            printf("\t Compare keys: %s\n", compare ? "FAIL" : "PASS");
            AssertEquals(0, compare);

        

            double dTimeSecs = 1.0e-3 * elapsedTime ;
            printf("Sorting Network, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u elements, NumDevsUsed = %u\n",
            (1.0e-6 * (double)N/dTimeSecs), dTimeSecs , N, 1);

            // Cleanup
            if (h_data) delete[] h_data;
            if (h_reference_data) delete[] h_reference_data;
            if (d_data) CUDA_SAFE_CALL(hipFree(d_data));

            hipEventDestroy(start);
            hipEventDestroy(stop);


            timetemp[iter] = elapsedTime;
            speedtemp[iter] = 1.0e-6 * (double)N/dTimeSecs;
        }
    
        timesum = 0;
        speedsum = 0;
        for (int iter = 0; iter < maxiter; iter++){
          timesum = timesum + timetemp[iter];
          speedsum = speedsum + speedtemp[iter];
        }
    
        time[n] = (float)timesum / maxiter;
        speed[n] = (float)speedsum / maxiter;
      }
    
    
    
    
      printf ("\n\n\tn:\n");
      for (n = minn; n <= maxn; n++){
        printf ("%d,\t", n);
      }
    
      printf ("\n\n\tN:\n");
      for (n = minn; n <= maxn; n++){
        printf ("%u,\t", (uint)pow(2,n));
      }
      
      printf ("\n\n\ttime:\n");
      for (n = minn; n <= maxn; n++){
        printf ("%lf,\t", time[n]);
      }
    
      printf ("\n\n");
    
      printf ("\n\n\tspeed:\n");
      for (n = minn; n <= maxn; n++){
        printf ("%lf,\t", speed[n]);
      }    
    
      printf ("\n\n");
    
    
}